/*
 * nvcc -std=c++17 -O2 -g 01_hello_world.cu -o 01_hello_world
 */

#include <cmath>
#include <cstdio>

#include <hip/hip_runtime.h>

__device__
void print_sqrt(double x) {
  printf("The square root of %g is %g\n", x, std::sqrt(x));
}

__global__
void kernel() {
  print_sqrt(2.);
}

int main() {
  kernel<<<1, 1>>>();
  hipDeviceSynchronize();
}
